#include "PyFRData.h"

#include <algorithm>
#include <cassert>
#include <cmath>
#include <iostream>
#include <iterator>
#include <limits>
#include <sstream>
#include <stdexcept>

#include <vtkm/CellShape.h>
#include <vtkm/CellTraits.h>
#include <vtkm/TopologyElementTag.h>
#include <vtkm/cont/CellSetSingleType.h>
#include <vtkm/cont/CoordinateSystem.h>
#include <vtkm/cont/DeviceAdapter.h>
#include <vtkm/cont/DataSet.h>
#include <vtkm/cont/Field.h>
#include <vtkm/cont/cuda/DeviceAdapterCuda.h>

#include <vtkm/VectorAnalysis.h>
#include <vtkm/worklet/WorkletMapField.h>
#include <vtkm/worklet/DispatcherMapField.h>

#include "ArrayHandleExposed.h"

namespace vtkm {
namespace worklet {



struct DeRho : public vtkm::worklet::WorkletMapField
{
  typedef void ControlSignature(
                                FieldIn<Scalar> rho,
                                FieldInOut<Scalar> u,
                                FieldInOut<Scalar> v,
                                FieldInOut<Scalar> w
                                );
  typedef void ExecutionSignature(_1, _2, _3, _4, WorkIndex);

  template<typename T>
  VTKM_EXEC_EXPORT void operator()(const T& rho,
                                   T& u,
                                   T& v,
                                   T& w,
                                   vtkm::Id index) const
  {
    u = u / rho;
    v = v / rho;
    w = w / rho;
  }
};

struct ComputeMagnitude : public vtkm::worklet::WorkletMapField
{
  typedef void ControlSignature(
                                FieldIn<Scalar> u,
                                FieldIn<Scalar> v,
                                FieldIn<Scalar> w,
                                FieldOut<Scalar> mag
                                );
  typedef void ExecutionSignature(_1, _2, _3, _4, WorkIndex);

  template<typename T>
  VTKM_EXEC_EXPORT void operator()(const T& u,
                                   const T& v,
                                   const T& w,
                                   T& mag,
                                   vtkm::Id index) const
  {
    mag = vtkm::Magnitude(vtkm::make_Vec(u,v,w));
  }
};

}
}

//------------------------------------------------------------------------------
std::map<int,std::string> PyFRData::fieldName;
std::map<std::string,int> PyFRData::fieldIndex;
bool PyFRData::mapsPopulated = PyFRData::PopulateMaps();

//------------------------------------------------------------------------------
bool PyFRData::PopulateMaps()
{
  fieldName[0] = "density";
  fieldName[1] = "pressure";
  fieldName[2] = "velocity_u";
  fieldName[3] = "velocity_v";
  fieldName[4] = "velocity_w";
  fieldName[5] = "velocity_magnitude";
  fieldName[6] = "density_gradient_magnitude";
  fieldName[7] = "pressure_gradient_magnitude";
  fieldName[8] = "velocity_gradient_magnitude";
  fieldName[9] = "velocity_qcriterion";

  for (unsigned i=0;i<10;i++)
    fieldIndex[fieldName[i]] = i;

  return true;
}

//------------------------------------------------------------------------------
PyFRData::PyFRData() : catalystData(NULL)
{
  this->bg_color[0] = this->bg_color[1] = this->bg_color[2] = NAN;
  this->image_size[0] = this->image_size[1] = 400u;
}

//------------------------------------------------------------------------------
PyFRData::~PyFRData()
{
}

//------------------------------------------------------------------------------
void PyFRData::Init(void* data)
{
  this->catalystData = static_cast<struct CatalystData*>(data);

  // we only take data from the first stored cell type (i.e. hexahedra)
  MeshDataForCellType* meshData = &(this->catalystData->meshData[0]);
  SolutionDataForCellType* solutionData =
    &(this->catalystData->solutionData[0]);
  this->isovals.resize(this->catalystData->niso);
  std::copy(this->catalystData->isovalues,
            this->catalystData->isovalues+this->catalystData->niso,
            this->isovals.begin());
  std::copy(this->catalystData->eye, this->catalystData->eye+3, this->eye);
  std::copy(this->catalystData->ref, this->catalystData->ref+3, this->ref);
  std::copy(this->catalystData->vup, this->catalystData->vup+3, this->vup);

  typedef ::vtkm::cont::DeviceAdapterTagCuda CudaTag;

  Vec3ArrayHandle vertices;
    {
    const vtkm::Vec<FPType,3> *vecData =
      reinterpret_cast<const vtkm::Vec<FPType,3>*>(meshData->vertices);
    typedef vtkm::cont::internal::Storage<vtkm::Vec<FPType,3>,
                                       vtkm::cont::StorageTagBasic> Vec3Storage;
    Vec3ArrayHandle tmp =
      Vec3ArrayHandle(Vec3Storage(vecData,
                                  meshData->nCells*meshData->nVerticesPerCell));
    vtkm::cont::DeviceAdapterAlgorithm<CudaTag>().
      Copy(tmp, vertices);
    }

  vtkm::cont::ArrayHandle<vtkm::Id> connectivity;
    {
    vtkm::cont::ArrayHandle<int32_t> tmp =
      vtkm::cont::make_ArrayHandle(meshData->con,
                                   (meshData->nSubdividedCells*
                   vtkm::CellTraits<vtkm::CellShapeTagHexahedron>::NUM_POINTS));
    vtkm::cont::ArrayHandleCast<vtkm::Id,
      vtkm::cont::ArrayHandle<int32_t> > cast(tmp);
    vtkm::cont::DeviceAdapterAlgorithm<CudaTag>().
      Copy(cast, connectivity);
    }

  vtkm::cont::CellSetSingleType<> cset(vtkm::CellShapeTagHexahedron(),
                                       meshData->nCells*meshData->nVerticesPerCell,
                                       "cells");
  cset.Fill(connectivity);

  StridedDataFunctor stridedDataFunctor[5];
  for (unsigned i=0;i<5;i++)
    {
    stridedDataFunctor[i].NumberOfCells = meshData->nCells;
    stridedDataFunctor[i].NVerticesPerCell = meshData->nVerticesPerCell;
    stridedDataFunctor[i].NSolutionTypes = 5;
    stridedDataFunctor[i].SolutionType = i;
    stridedDataFunctor[i].CellStride = solutionData->lsdim;
    stridedDataFunctor[i].VertexStride = solutionData->ldim;
    }

  RawDataArrayHandle rawSolutionArray = vtkm::cont::cuda::make_ArrayHandle(
    static_cast<FPType*>(solutionData->solution),
    solutionData->ldim*meshData->nVerticesPerCell);

  DataIndexArrayHandle densityIndexArray(stridedDataFunctor[0],
                                   meshData->nCells*meshData->nVerticesPerCell);
  CatalystMappedDataArrayHandle densityArray(densityIndexArray, rawSolutionArray);

  DataIndexArrayHandle velocity_uIndexArray(stridedDataFunctor[1],
                                   meshData->nCells*meshData->nVerticesPerCell);
  CatalystMappedDataArrayHandle velocity_uArray(velocity_uIndexArray, rawSolutionArray);

  DataIndexArrayHandle velocity_vIndexArray(stridedDataFunctor[2],
                                   meshData->nCells*meshData->nVerticesPerCell);
  CatalystMappedDataArrayHandle velocity_vArray(velocity_vIndexArray, rawSolutionArray);

  DataIndexArrayHandle velocity_wIndexArray(stridedDataFunctor[3],
                                   meshData->nCells*meshData->nVerticesPerCell);
  CatalystMappedDataArrayHandle velocity_wArray(velocity_wIndexArray, rawSolutionArray);

  DataIndexArrayHandle pressureIndexArray(stridedDataFunctor[4],
                                   meshData->nCells*meshData->nVerticesPerCell);
  CatalystMappedDataArrayHandle pressureArray(pressureIndexArray, rawSolutionArray);

  //correct the velocity u,v,w arrays to be u,v,w instead of being
  //u*rho, v*rho, w*rho. Note this is all done inplace
  vtkm::worklet::DispatcherMapField< vtkm::worklet::DeRho, CudaTag > dispatchDeRho;
  dispatchDeRho.Invoke(densityArray, velocity_uArray, velocity_vArray, velocity_wArray);

  //compute the magnitude of the velocity.
  ScalarDataArrayHandle velocity_mArray;
  vtkm::worklet::DispatcherMapField< vtkm::worklet::ComputeMagnitude, CudaTag > dispatcher;
  dispatcher.Invoke(velocity_uArray, velocity_vArray, velocity_wArray, velocity_mArray);

  enum ElemType { CONSTANT=0, LINEAR=1, QUADRATIC=2 };
  vtkm::cont::Field density("density",LINEAR,vtkm::cont::Field::ASSOC_POINTS,vtkm::cont::DynamicArrayHandle(densityArray));
  vtkm::cont::Field velocity_u("velocity_u",LINEAR,vtkm::cont::Field::ASSOC_POINTS,vtkm::cont::DynamicArrayHandle(velocity_uArray));
  vtkm::cont::Field velocity_v("velocity_v",LINEAR,vtkm::cont::Field::ASSOC_POINTS,vtkm::cont::DynamicArrayHandle(velocity_vArray));
  vtkm::cont::Field velocity_w("velocity_w",LINEAR,vtkm::cont::Field::ASSOC_POINTS,vtkm::cont::DynamicArrayHandle(velocity_wArray));
  vtkm::cont::Field velocity_m("velocity_magnitude",LINEAR,vtkm::cont::Field::ASSOC_POINTS,vtkm::cont::DynamicArrayHandle(velocity_mArray));
  vtkm::cont::Field pressure("pressure",LINEAR,vtkm::cont::Field::ASSOC_POINTS,vtkm::cont::DynamicArrayHandle(pressureArray));

  this->dataSet.AddCoordinateSystem(vtkm::cont::CoordinateSystem("coordinates",
                                                                 1,vertices));

  this->dataSet.AddField(density);
  this->dataSet.AddField(pressure);
  this->dataSet.AddField(velocity_u);
  this->dataSet.AddField(velocity_v);
  this->dataSet.AddField(velocity_w);
  this->dataSet.AddField(velocity_m);
  this->dataSet.AddCellSet(cset);
}

namespace {

PyFRData::CatalystMappedDataArrayHandle
make_CatalystHandle(const vtkm::cont::Field& field)
{
  return field.GetData().CastToArrayHandle(
    PyFRData::CatalystMappedDataArrayHandle::ValueType(),
    PyFRData::CatalystMappedDataArrayHandle::StorageTag());
}

PyFRData::ScalarDataArrayHandle
make_ScalarHandle(const vtkm::cont::Field& field)
{
  return field.GetData().CastToArrayHandle(
      PyFRData::ScalarDataArrayHandle::ValueType(),
      PyFRData::ScalarDataArrayHandle::StorageTag());
}

}

//------------------------------------------------------------------------------
void PyFRData::Update()
{
  typedef ::vtkm::cont::DeviceAdapterTagCuda CudaTag;


  vtkm::cont::Field densField = this->dataSet.GetField("density");
  vtkm::cont::Field veluField = this->dataSet.GetField("velocity_u");
  vtkm::cont::Field velvField = this->dataSet.GetField("velocity_v");
  vtkm::cont::Field velwField = this->dataSet.GetField("velocity_w");
  vtkm::cont::Field velmField = this->dataSet.GetField("velocity_magnitude");

  PyFRData::CatalystMappedDataArrayHandle dens = make_CatalystHandle(densField);
  PyFRData::CatalystMappedDataArrayHandle velu = make_CatalystHandle(veluField);
  PyFRData::CatalystMappedDataArrayHandle velv = make_CatalystHandle(velvField);
  PyFRData::CatalystMappedDataArrayHandle velw = make_CatalystHandle(velwField);
  PyFRData::ScalarDataArrayHandle velm = make_ScalarHandle(velmField);

  //correct the velocity u,v,w arrays to be u,v,w instead of being
  //u*rho, v*rho, w*rho. Note this is all done inplace
  vtkm::worklet::DispatcherMapField< vtkm::worklet::DeRho, CudaTag > dispatchDeRho;
  dispatchDeRho.Invoke(dens, velu, velv, velw);

  //compute the magnitude of the velocity.
  vtkm::worklet::DispatcherMapField< vtkm::worklet::ComputeMagnitude, CudaTag > dispatcher;
  dispatcher.Invoke(velu, velv, velw, velm);

}

bool PyFRData::PrintMetadata() const {
  return this->catalystData->metadata;
}
