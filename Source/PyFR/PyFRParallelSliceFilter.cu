#include "PyFRParallelSliceFilter.h"

#include <vtkm/ImplicitFunctions.h>
#include <vtkm/cont/cuda/DeviceAdapterCuda.h>

#include "CrinkleClip.h"
#include "IsosurfaceHexahedra.h"
#include "PyFRData.h"
#include "PyFRContour.h"
#include "PyFRContourData.h"

//----------------------------------------------------------------------------
PyFRParallelSliceFilter::PyFRParallelSliceFilter() : NPlanes(1), Spacing(1.)
{
  this->Origin[0] = this->Origin[1] = this->Origin[2] = 0.;
  this->Normal[0] = this->Normal[1] = 0.;
  this->Normal[2] = 1.;
}

//----------------------------------------------------------------------------
PyFRParallelSliceFilter::~PyFRParallelSliceFilter()
{
}

//----------------------------------------------------------------------------
void PyFRParallelSliceFilter::SetPlane(FPType origin_x,
                                       FPType origin_y,
                                       FPType origin_z,
                                       FPType normal_x,
                                       FPType normal_y,
                                       FPType normal_z)
{
  this->Origin[0] = origin_x;
  this->Origin[1] = origin_y;
  this->Origin[2] = origin_z;
  this->Normal[0] = normal_x;
  this->Normal[1] = normal_y;
  this->Normal[2] = normal_z;
}

//----------------------------------------------------------------------------
void PyFRParallelSliceFilter::operator()(PyFRData* input,
                                         PyFRContourData* output)
{
  typedef PyFRData::Vec3ArrayHandle CoordinateArrayHandle;
  typedef std::vector<vtkm::cont::ArrayHandle<vtkm::Vec<FPType,3> > >
    Vec3HandleVec;
  typedef std::vector<FPType> DataVec;
  typedef PyFRData::CellSet CellSet;

  const vtkm::cont::DataSet& dataSet = input->GetDataSet();

  CoordinateArrayHandle coords = dataSet.GetCoordinateSystem().GetData()
    .CastToArrayHandle(CoordinateArrayHandle::ValueType(),
                       CoordinateArrayHandle::StorageTag());

  vtkm::Plane func(vtkm::Vec<FPType,3>(this->Origin[0],
                                       this->Origin[1],
                                       this->Origin[2]),
                   vtkm::Vec<FPType,3>(this->Normal[0],
                                       this->Normal[1],
                                       this->Normal[2]));

  vtkm::ImplicitFunctionValue<vtkm::Plane> function(func);

  vtkm::cont::ArrayHandleTransform<FPType,CoordinateArrayHandle,
    vtkm::ImplicitFunctionValue<vtkm::Plane> > dataArray(coords,function);

  DataVec dataVec;
  Vec3HandleVec verticesVec;
  Vec3HandleVec normalsVec;
  output->SetNumberOfContours(this->NPlanes);
  for (unsigned i=0;i<output->GetNumberOfContours();i++)
    {
    dataVec.push_back(i*this->Spacing);
    verticesVec.push_back(output->GetContour(i).GetVertices());
    normalsVec.push_back(output->GetContour(i).GetNormals());
    }

  vtkm::cont::Timer<CudaTag> timer;
  isosurfaceFilter.Run(dataVec,
                       dataSet.GetCellSet().CastTo(CellSet()),
                       dataSet.GetCoordinateSystem(),
                       dataArray,
                       verticesVec,
                       normalsVec);
}

//----------------------------------------------------------------------------
void PyFRParallelSliceFilter::MapFieldOntoSlices(int field,
                                                 PyFRData* input,
                                                 PyFRContourData* output)
{
  std::cout << "Coloring slice with field: " << PyFRData::FieldName(field) << std::endl;
  typedef std::vector<PyFRContour::ScalarDataArrayHandle> ScalarDataHandleVec;

  const vtkm::cont::DataSet& dataSet = input->GetDataSet();

  ScalarDataHandleVec scalarDataHandleVec;
  for (unsigned j=0;j<output->GetNumberOfContours();j++)
    {
    output->GetContour(j).SetScalarDataType(field);
    PyFRContour::ScalarDataArrayHandle scalars_out =
      output->GetContour(j).GetScalarData();
    scalarDataHandleVec.push_back(scalars_out);
    }

  vtkm::cont::Field projectedField =
    dataSet.GetField(PyFRData::FieldName(field));

  PyFRData::ScalarDataArrayHandle projectedArray = projectedField.GetData()
    .CastToArrayHandle(PyFRData::ScalarDataArrayHandle::ValueType(),
                       PyFRData::ScalarDataArrayHandle::StorageTag());

  isosurfaceFilter.MapFieldOntoIsosurfaces<
    PyFRData::ScalarDataArrayHandle,
      PyFRContour::ScalarDataArrayHandle>(projectedArray,
                                          scalarDataHandleVec);
}
