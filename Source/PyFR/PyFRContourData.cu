#include "hip/hip_runtime.h"
#include <algorithm>
#include <stdexcept>
#include "PyFRContourData.h"

#include "PyFRContour.h"
#include "ColorTable.h"
#include "Bounds.h"

#include <vtkm/Math.h>
#include <vtkm/Pair.h>
#include <vtkm/Types.h>
#include <vtkm/VectorAnalysis.h>

#include <vtkm/cont/cuda/DeviceAdapterCuda.h>
#include <vtkm/cont/DeviceAdapter.h>
#include <vtkm/cont/ArrayHandleTransform.h>

#include <vtkm/cont/ArrayHandleCast.h>
#include <vtkm/opengl/TransferToOpenGL.h>
#include <vtkm/opengl/cuda/internal/TransferToOpenGL.h>


class PyFRContourData::ContourDataImpl
{
public:
  ContourDataImpl()
  {
    TablePreset = ColorTable::GRAYSCALE;
    this->Table = make_ColorTable(TablePreset,0.0,1.0);
  }

  ColorTable::Preset TablePreset;

  RuntimeColorTable Table;
  std::vector<PyFRContour> Contours;
};

//----------------------------------------------------------------------------
PyFRContourData::PyFRContourData()
{
  this->Impl = new ContourDataImpl;
}

//----------------------------------------------------------------------------
PyFRContourData::~PyFRContourData()
{
  this->Impl->Table.ReleaseResources();
  delete this->Impl;
}

//----------------------------------------------------------------------------
unsigned PyFRContourData::GetNumberOfContours() const
{ return this->Impl->Contours.size(); }
//----------------------------------------------------------------------------
PyFRContour& PyFRContourData::GetContour(int i)
{ return this->Impl->Contours[i]; }
//----------------------------------------------------------------------------
const PyFRContour& PyFRContourData::GetContour(int i) const
{ return this->Impl->Contours[i]; }

//----------------------------------------------------------------------------
void PyFRContourData::SetNumberOfContours(unsigned nContours)
{
  // NB: Cannot call resize to increase the lengths of vectors of array
  // handles (or classes containing them)! You will end up with a vector of
  // smart pointers to the same array instance. A specialization of
  // std::allocator<> for array handles should be created.

  for (unsigned i=this->Impl->Contours.size();i<nContours;i++)
    {
    this->Impl->Contours.push_back(PyFRContour(this->Impl->Table));
    }
  unsigned contourSize = this->Impl->Contours.size();
  for (unsigned i=nContours;i<contourSize;i++)
    {
    this->Impl->Contours.pop_back();
    }
}

//----------------------------------------------------------------------------
unsigned PyFRContourData::GetContourSize(int contour) const
{
  if(contour < this->Impl->Contours.size())
    return this->GetContour(contour).GetVertices().GetNumberOfValues();
  return 0;
}

//----------------------------------------------------------------------------
void PyFRContourData::ComputeContourBounds(int contour,FPType* bounds) const
{
  typedef ::vtkm::cont::DeviceAdapterTagCuda CudaTag;
  typedef vtkm::cont::DeviceAdapterAlgorithm<CudaTag> Algorithm;
  typedef vtkm::Vec<vtkm::Float64, 3> ResultType;
  typedef vtkm::Pair<ResultType, ResultType> MinMaxPairType;
  typedef PyFRContour::Vec3ArrayHandle ArrayHandleType;

  MinMaxPairType initialValue =
    make_Pair(ResultType(vtkm::Infinity64()),
              ResultType(vtkm::NegativeInfinity64()));

  vtkm::cont::ArrayHandleTransform<MinMaxPairType, ArrayHandleType,
    internal::InputToOutputTypeTransform<3> > input(this->GetContour(contour)
                                                    .GetVertices());

  MinMaxPairType result = Algorithm::Reduce(input, initialValue,
                                            internal::MinMax<3>());

  for (unsigned i=0;i<3;i++)
    {
    bounds[2*i] = result.first[i];
    bounds[2*i+1] = result.second[i];
    }
}

//----------------------------------------------------------------------------
void PyFRContourData::ComputeBounds(FPType* bounds) const
{
  for (unsigned i=0;i<3;i++)
    {
    bounds[2*i] = std::numeric_limits<FPType>::max();
    bounds[2*i+1] = std::numeric_limits<FPType>::min();
    }
  for (unsigned i=0;i<this->GetNumberOfContours();i++)
    {
    FPType b[6];
    this->ComputeContourBounds(i,b);
    bounds[0] = std::min(bounds[0], b[0]);
    bounds[1] = std::max(bounds[1], b[1]);
    bounds[2] = std::min(bounds[2], b[2]);
    bounds[3] = std::max(bounds[3], b[3]);
    bounds[4] = std::min(bounds[4], b[4]);
    bounds[5] = std::max(bounds[5], b[5]);
    }
}

//----------------------------------------------------------------------------
void PyFRContourData::SetColorPalette(int preset, FPType min, FPType max)
{
  this->Impl->Table = make_ColorTable(static_cast<ColorTable::Preset>(preset), min, max);
  this->Impl->TablePreset = static_cast<ColorTable::Preset>(preset);

  for (unsigned i=0;i<this->GetNumberOfContours();i++)
    {
    this->Impl->Contours[i].ChangeColorTable(this->Impl->Table);
    }
}


//----------------------------------------------------------------------------
void PyFRContourData::SetColorPreset(int preset)
{
  this->SetColorPalette(static_cast<ColorTable::Preset>(preset),
                        this->Impl->Table.Min,
                        this->Impl->Table.Max);
}

//----------------------------------------------------------------------------
void PyFRContourData::SetColorRange(FPType min,FPType max)
{
  this->SetColorPalette(this->Impl->TablePreset,
                        min,
                        max);
}

namespace transfer
{

typedef ::vtkm::cont::DeviceAdapterTagCuda CudaTag;

//----------------------------------------------------------------------------
template<typename HandleType>
void to_gl(vtkm::Vec<vtkm::Float64,3>, const HandleType& handle, unsigned int& glHandle)
{
  //make an implicit wrapper to float32 around the float64 array
  vtkm::cont::ArrayHandleCast<vtkm::Vec<vtkm::Float32,3>,HandleType> asF32 =
    vtkm::cont::make_ArrayHandleCast(handle, vtkm::Vec<vtkm::Float32,3>());

  //transfer the array to openGL now as a float32 array
  vtkm::opengl::TransferToOpenGL(asF32, glHandle, CudaTag());
}

//----------------------------------------------------------------------------
template<typename HandleType>
void to_gl(vtkm::Float32, const HandleType& handle, unsigned int& glHandle)
{
  vtkm::opengl::TransferToOpenGL(handle, glHandle, CudaTag());
}

//----------------------------------------------------------------------------
void coords(PyFRContourData* data, int index, unsigned int& glHandle)
{
  to_gl(FPType(), data->GetContour(index).GetVertices(), glHandle);
}

//----------------------------------------------------------------------------
void normals(PyFRContourData* data, int index, unsigned int& glHandle)
{
  to_gl(FPType(), data->GetContour(index).GetNormals(), glHandle);
}

//----------------------------------------------------------------------------
void colors(PyFRContourData* data, int index, unsigned int& glHandle)
{
  //no need to worry about conversion, since this is always Vec4 of uint8's
  vtkm::opengl::TransferToOpenGL( data->GetContour(index).GetColorData(),
                                  glHandle,
                                  CudaTag());
}

} //namespace transfer
